// #include <math.h>

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>


// Constants and parameters
const double length_x = 1.0;
const double length_y = 0.6;
const int nx = 200;
const int ny = 120;
const double dx = length_x / nx;
const double dy = length_y / ny;
const double dt = 0.000001;
const double nu = 0.5;
const double div_tolerance = 1e-3;
const double beta0 = 1.0;
const int max_iterations = 200;
const double total_time = 0.0005;
const double beta = beta0 / (2*dt*(1 / (dx*dx) + 1 / (dy*dy)));

const int max_number_of_frames = 50;

#define s_d sizeof(double)
#define s_i sizeof(int)
#define s_i16 sizeof(uint16_t)

// CUDA error checking macro
#define CUDA_CHECK_ERROR(call)                                             \
    {                                                                      \
        hipError_t err = call;                                            \
        if (err != hipSuccess) {                                          \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(err));                    \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

// Host velocity fields, pressure, and divergence
double **h_u;                   // x-velocity on cell faces (ny x nx+1)
double **h_v;                   // y-velocity on cell faces (ny+1 x nx)
double **h_p;                   // pressure at cell centers (ny x nx)
double **h_div;                 // divergence (ny x nx)
double **h_u_center;            // x-velocity at cell centers (ny x nx)
double **h_v_center;            // y-velocity at cell centers (ny x nx)
double **h_velocity_magnitude;  // Velocity magnitude (ny x nx)
int **h_mask;                   // Obstacle mask (ny x nx)
int *h_converged;               // Convergence flag

// Device 1D arrays
double *d_u;                   // x-velocity on cell faces
double *d_v;                   // y-velocity on cell faces
double *d_p;                   // pressure at cell centers
double *d_div;                 // divergence
double *d_u_next;              // temporary x-velocity
double *d_v_next;              // temporary y-velocity
double *d_u_center;            // x-velocity at cell centers
double *d_v_center;            // y-velocity at cell centers
double *d_velocity_magnitude;  // Velocity magnitude
int *d_mask;                   // Obstacle mask
int *d_converged;              // Convergence flag

// Obstacle definition
const int obstacle_x_start = (int)(nx*0.3);
const int obstacle_x_end = (int)(nx*0.4);
const int obstacle_y_start = (int)(ny*0);
const int obstacle_y_end = (int)(ny*0.9);

// Function to convert float to half-precision float
uint16_t float_to_half(float f) {
    // Convert float to bits
    uint32_t f_bits = *((uint32_t *)&f);

    // Extract sign, mantissa and exponent
    uint16_t sign = (f_bits >> 31) & 0x01;
    uint16_t exp = (f_bits >> 23) & 0xFF;
    uint32_t mant = f_bits & 0x7FFFFF;

    // Handle special cases
    if (f == 0.0f) return 0;
    if (exp == 0xFF && mant == 0) return (sign << 15) | 0x7C00;  // Infinity
    if (exp == 0xFF && mant != 0) return (sign << 15) | 0x7E00;  // NaN

    // Calculate exponent for half precision
    int16_t half_exp = exp - 127 + 15;

    // Handle normal numbers
    if (half_exp >= 1 && half_exp <= 30) {
        // Normal number
        mant = mant >> 13;
        return (sign << 15) | (half_exp << 10) | mant;
    } else if (half_exp <= 0 && half_exp >= -10) {
        // Subnormal number (denormalized)
        // Shift mantissa to account for the exponent difference
        // Include the implied leading 1 bit for the mantissa
        mant = (mant | 0x800000) >> (14 - half_exp);
        return (sign << 15) | mant;
    } else if (half_exp > 30) {
        // Overflow to infinity
        return (sign << 15) | 0x7C00;
    } else {
        // Underflow to zero
        return (sign << 15);
    }
}

// Function to allocate memory for 2D arrays on host
double **allocate_2d_array(int rows, int cols) {
    double **arr = (double **)malloc(rows*sizeof(double *));
    double *data = (double *)calloc(rows*cols, s_d);
    for (int i = 0; i < rows; i++) {
        arr[i] = &data[i*cols];
    }
    return arr;
}

// Function to allocate memory for 2D integer arrays on host
int **allocate_2d_int_array(int rows, int cols) {
    int **arr = (int **)malloc(rows*sizeof(int *));
    int *data = (int *)malloc(rows*cols*s_i);
    for (int i = 0; i < rows; i++) {
        arr[i] = &data[i*cols];
        // Initialize with ones
        for (int j = 0; j < cols; j++) {
            arr[i][j] = 1;
        }
    }
    return arr;
}

// Function to free memory for 2D arrays on host
void free_2d_array(double **arr) {
    free(arr[0]);  // Free the data block
    free(arr);     // Free the pointers
}

// Function to free memory for 2D integer arrays on host
void free_2d_int_array(int **arr) {
    free(arr[0]);  // Free the data block
    free(arr);     // Free the pointers
}

// Function to copy 2D host arrays to 1D device arrays
void copy_host_to_device() {
    // Copy u
    for (int j = 0; j < ny; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_u + j*(nx+1), h_u[j], (nx+1)*s_d,
                                    hipMemcpyHostToDevice));
    }

    // Copy v
    for (int j = 0; j < ny+1; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_v + j*nx, h_v[j], nx*s_d,
                                    hipMemcpyHostToDevice));
    }

    // Copy p
    for (int j = 0; j < ny; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_p + j*nx, h_p[j], nx*s_d,
                                    hipMemcpyHostToDevice));
    }

    // Copy mask
    for (int j = 0; j < ny; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_mask + j*nx, h_mask[j], nx*s_i,
                                    hipMemcpyHostToDevice));
    }
}

// Function to copy 1D device arrays to 2D host arrays
void copy_device_to_host() {
    // // Copy u
    // for (int j = 0; j < ny; j++) {
    //     CUDA_CHECK_ERROR(cudaMemcpy(h_u[j], d_u + j*(nx+1),
    //                                 (nx+1)*s_d,
    //                                 cudaMemcpyDeviceToHost));
    // }

    // // Copy v
    // for (int j = 0; j < ny+1; j++) {
    //     CUDA_CHECK_ERROR(cudaMemcpy(h_v[j], d_v + j*nx, nx *
    //     s_d,
    //                                 cudaMemcpyDeviceToHost));
    // }

    // // Copy p
    // for (int j = 0; j < ny; j++) {
    //     CUDA_CHECK_ERROR(cudaMemcpy(h_p[j], d_p + j*nx, nx *
    //     s_d,
    //                                 cudaMemcpyDeviceToHost));
    // }

    // Copy center velocities and magnitude
    for (int j = 0; j < ny; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(h_u_center[j], d_u_center + j*nx,
                                    nx*s_d,
                                    hipMemcpyDeviceToHost));
        CUDA_CHECK_ERROR(hipMemcpy(h_v_center[j], d_v_center + j*nx,
                                    nx*s_d,
                                    hipMemcpyDeviceToHost));
        CUDA_CHECK_ERROR(hipMemcpy(h_velocity_magnitude[j], d_velocity_magnitude + j*nx,
                                    nx*s_d,
                                    hipMemcpyDeviceToHost));
    }
}

// CUDA kernel to apply boundary conditions
__global__ void apply_boundary_conditions_kernel(double *u, double *v,
                                                 int *mask, int nx, int ny) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    // Left boundary (inlet): constant velocity
    if (i == 0 && j < ny) {
        u[j*(nx+1) + 0] = 1.0;
    }

    // Right boundary (outlet): zero gradient
    if (i == nx && j < ny) {
        u[j*(nx+1) + nx] = u[j*(nx+1) + nx-1];
    }

    // Top and bottom boundaries: no-slip condition
    if (i < nx && j == 0) {
        v[0*nx + i] = 0.0;
    }

    if (i < nx && j == ny) {
        v[ny*nx + i] = 0.0;
    }

    if (i < nx+1 && j == 0) {
        u[0*(nx+1) + i] = 0.0;
    }

    if (i < nx+1 && j == ny-1) {
        u[(ny-1)*(nx+1) + i] = 0.0;
    }

    // Synchronize threads
    __syncthreads();

    // Obstacle boundary conditions: no-slip condition
    if (i < nx && j < ny) {
        if (mask[j*nx + i] == 0) {
            u[j*(nx+1) + i] = 0.0;
            u[j*(nx+1) + i+1] = 0.0;
            v[j*nx + i] = 0.0;
            v[(j+1)*nx + i] = 0.0;
        }
    }

    // Velocity clamping
    if (i < nx+1 && j < ny) {
        if (u[j*(nx+1) + i] > 5.0) u[j*(nx+1) + i] = 5.0;
        if (u[j*(nx+1) + i] < -5.0) u[j*(nx+1) + i] = -5.0;
    }

    if (i < nx && j < ny+1) {
        if (v[j*nx + i] > 5.0) v[j*nx + i] = 5.0;
        if (v[j*nx + i] < -5.0) v[j*nx + i] = -5.0;
    }
}

// CUDA kernel to compute the divergence in each cell
__global__ void compute_divergence_kernel(double *u, double *v, double *div,
                                          int nx, int ny, double dx,
                                          double dy) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        div[j*nx + i] = (u[j*(nx+1) + i+1] - u[j*(nx+1) + i]) / dx +
                          (v[(j+1)*nx + i] - v[j*nx + i]) / dy;
    }
}

// CUDA kernel to update velocities based on Navier-Stokes equation
__global__ void update_velocities_kernel(double *u, double *v, double *p,
                                         double *u_next, double *v_next, int nx,
                                         int ny, double dx, double dy,
                                         double dt, double nu) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    // Internal cells for u (excluding boundaries)
    if (i >= 1 && i < nx && j >= 1 && j < ny-1) {
        // Calculate averages for u at cell centers
        double u_i__j = 0.5*(u[j*(nx+1) + i-1] + u[j*(nx+1) + i]);
        double u_i_plus1__j =
            0.5*(u[j*(nx+1) + i] + u[j*(nx+1) + i+1]);

        // Calculate uv product terms (Eq. uv-average)
        double uv_i_plushalf__j_minushalf =
            0.25*(u[(j-1)*(nx+1) + i] + u[j*(nx+1) + i]) *
            (v[j*nx + i-1] + v[j*nx + i]);
        double uv_i_plushalf__j_plushalf =
            0.25*(u[j*(nx+1) + i] + u[(j+1)*(nx+1) + i]) *
            (v[(j+1)*nx + i-1] + v[(j+1)*nx + i]);

        // Implement u-momentum equation (Eq. u-finite-diff)
        u_next[j*(nx+1) + i] =
            u[j*(nx+1) + i] + dt * ( 
                + (u_i__j*u_i__j - u_i_plus1__j*u_i_plus1__j) / dx
                + (uv_i_plushalf__j_minushalf - uv_i_plushalf__j_plushalf) / dy +
                + (p[j*nx + i-1] - p[j*nx + i]) / dx 
                + nu*(
                    (u[j*(nx+1) + i+1] - 2*u[j*(nx+1) + i] + u[j*(nx+1) + i-1]) / (dx*dx)
                    + (u[(j+1)*(nx+1) + i] - 2*u[j*(nx+1) + i] + u[(j-1)*(nx+1) + i]) / (dy*dy)
                )
            );
    }

    // Internal cells for v (excluding boundaries)
    if (i >= 1 && i < nx-1 && j >= 1 && j < ny) {
        // Calculate averages for v at cell centers
        double v_i__j = 0.5*(v[(j-1)*nx + i] + v[j*nx + i]);
        double v_i__j_plus1 = 0.5*(v[j*nx + i] + v[(j+1)*nx + i]);

        // Calculate uv product terms (Eq. uv-average)
        double uv_i_minushalf__j_plushalf =
            0.25*(u[(j-1)*(nx+1) + i] + u[j*(nx+1) + i]) *
            (v[j*nx + i-1] + v[j*nx + i]);
        double uv_i_plushalf__j_plushalf =
            0.25*(u[(j-1)*(nx+1) + i+1] + u[j*(nx+1) + i+1]) *
            (v[j*nx + i] + v[j*nx + i+1]);

        // Implement v-momentum equation (Eq. v-finite-diff)
        v_next[j*nx + i] =
            v[j*nx + i] + dt * ( 
                + (v_i__j*v_i__j - v_i__j_plus1*v_i__j_plus1) / dy
                + (uv_i_minushalf__j_plushalf - uv_i_plushalf__j_plushalf) / dx +
                + (p[(j-1)*nx + i] - p[j*nx + i]) / dy 
                + nu*(
                    (v[j*nx + i+1] - 2*v[j*nx + i] + v[j*nx + i-1]) / (dx*dx)
                    + (v[(j+1)*nx + i] - 2*v[j*nx + i] + v[(j-1)*nx + i]) / (dy*dy)
                )
            );
    }
}

// CUDA kernel to copy the updated velocities back to the main velocity arrays
__global__ void update_velocities_final_kernel(double *u, double *v,
                                               double *u_next, double *v_next,
                                               int nx, int ny) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i >= 1 && i < nx && j >= 1 && j < ny-1) {
        u[j*(nx+1) + i] = u_next[j*(nx+1) + i];
    }

    if (i >= 1 && i < nx-1 && j >= 1 && j < ny) {
        v[j*nx + i] = v_next[j*nx + i];
    }
}

// CUDA kernel to perform pressure correction iterations
// __global__ void pressure_correction_kernel(double *u, double *v, double *p,
//                                            double *div, int *mask, int nx,
//                                            int ny, double dx, double dy,
//                                            double dt, double beta,
//                                            double div_tolerance,
//                                            int *converged) {
//     int i = blockIdx.x*blockDim.x + threadIdx.x;
//     int j = blockIdx.y*blockDim.y + threadIdx.y;

//     if (i < nx && j < ny) {
//         if (mask[j*nx + i] == 1 && fabs(div[j*nx + i]) > div_tolerance) {
//             atomicMin(converged, 0);

//             double delta_p = -beta*div[j*nx + i];
//             // Update pressure w/ pressure gradient damping
//             p[j*nx + i] += delta_p*0.7;

//             // Adjust velocity components
//             if (i+1 <= nx) u[j*(nx+1) + i+1] += 0.5*dt / dx*delta_p;
//             if (i >= 0) u[j*(nx+1) + i] -= 0.5*dt / dx*delta_p;
//             if (j+1 <= ny) v[(j+1)*nx + i] += 0.5*dt / dy*delta_p;
//             if (j >= 0) v[j*nx + i] -= 0.5*dt / dy*delta_p;
//         }
//     }
// }
__global__ void pressure_correction_kernel(double *u, double *v, double *p,
                                           double *div, int *mask, int nx,
                                           int ny, double dx, double dy,
                                           double dt, double beta,
                                           double div_tolerance,
                                           int *converged) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        if (mask[j*nx + i] == 1 && fabs(div[j*nx + i]) > div_tolerance) {
            atomicAnd(converged, 0);

            double delta_p = -beta*div[j*nx + i];
            p[j*nx + i] += delta_p * 0.7;

            if (i+1 <= nx) atomicAdd(&u[j*(nx+1) + i+1], 0.5*dt/dx * delta_p);
            if (i >= 0) atomicAdd(&u[j*(nx+1) + i], -0.5*dt/dx * delta_p);
            if (j+1 <= ny) atomicAdd(&v[(j+1)*nx + i], 0.5*dt/dy * delta_p);
            if (j >= 0) atomicAdd(&v[j*nx + i], -0.5*dt/dy * delta_p);
        }
    }
}

// CUDA kernel to calculate velocities at cell centers and magnitude
__global__ void calculate_center_velocities_and_magnitude_kernel(
    double *u, double *v, double *u_center, double *v_center,
    double *velocity_magnitude, int nx, int ny) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        u_center[j*nx + i] = 0.5*(u[j*(nx+1) + i] + u[j*(nx+1) + i+1]);
        v_center[j*nx + i] = 0.5*(v[j*nx + i] + v[(j+1)*nx + i]);
        velocity_magnitude[j*nx + i] =
            sqrt(u_center[j*nx + i]*u_center[j*nx + i] +
                 v_center[j*nx + i]*v_center[j*nx + i]);
    }
}

// Function to write the current state to the binary file
void write_state_to_binary(FILE *fp_u_center, FILE *fp_v_center,
                           FILE *fp_magnitude) {
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            uint16_t half_val = float_to_half((float)h_u_center[j][i]);
            fwrite(&half_val, s_i16, 1, fp_u_center);
        }
    }

    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            uint16_t half_val = float_to_half((float)h_v_center[j][i]);
            fwrite(&half_val, s_i16, 1, fp_v_center);
        }
    }

    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            uint16_t half_val =
                float_to_half((float)h_velocity_magnitude[j][i]);
            fwrite(&half_val, s_i16, 1, fp_magnitude);
        }
    }
}

// Function to write metadata to a JSON file
void write_metadata_to_json(const char *filename, double compute_time_seconds, double total_time_seconds) {
    FILE *fp = fopen(filename, "w");
    if (fp == NULL) {
        perror("Error opening file");
        return;
    }

    fprintf(fp, "{\n");
    fprintf(fp, "    \"length_x\": %f,\n", length_x);
    fprintf(fp, "    \"length_y\": %f,\n", length_y);
    fprintf(fp, "    \"nx\": %d,\n", nx);
    fprintf(fp, "    \"ny\": %d,\n", ny);
    fprintf(fp, "    \"dx\": %f,\n", dx);
    fprintf(fp, "    \"dy\": %f,\n", dy);
    fprintf(fp, "    \"dt\": %f,\n", dt);
    fprintf(fp, "    \"nu\": %f,\n", nu);
    fprintf(fp, "    \"div_tolerance\": %e,\n", div_tolerance);
    fprintf(fp, "    \"beta0\": %f,\n", beta0);
    fprintf(fp, "    \"max_iterations\": %d,\n", max_iterations);
    fprintf(fp, "    \"total_time\": %f,\n", total_time);
    fprintf(fp, "    \"beta\": %f,\n", beta);
    fprintf(fp, "    \"obstacle_x_start\": %d,\n", obstacle_x_start);
    fprintf(fp, "    \"obstacle_x_end\": %d,\n", obstacle_x_end);
    fprintf(fp, "    \"obstacle_y_start\": %d,\n", obstacle_y_start);
    fprintf(fp, "    \"obstacle_y_end\": %d,\n", obstacle_y_end);

    fprintf(fp, "    \"data_dtype\": \"float16\",\n");
    fprintf(fp, "    \"output_interval_in_c_steps\": %d,\n",
            (int)(total_time / dt) / max_number_of_frames);
    fprintf(fp, "    \"num_frames_output\": %d,\n",
            ((int)(total_time / dt) /
             ((int)(total_time / dt) / max_number_of_frames)));
    fprintf(fp, "    \"total_compute_time_seconds\": %f,\n",
            compute_time_seconds);
    fprintf(fp, "    \"total_time_seconds\": %f,\n",
            total_time_seconds);
    fprintf(fp, "    \"parallelization\": \"CUDA\"\n");

    fprintf(fp, "}");
    fclose(fp);
}

int main() {
    clock_t total_start_time = clock();
    double total_compute_time_seconds = 0.0;

    // Allocate host memory
    h_u = allocate_2d_array(ny, nx+1);
    h_v = allocate_2d_array(ny+1, nx);
    h_p = allocate_2d_array(ny, nx);
    h_div = allocate_2d_array(ny, nx);
    h_mask = allocate_2d_int_array(ny, nx);
    h_u_center = allocate_2d_array(ny, nx);
    h_v_center = allocate_2d_array(ny, nx);
    h_velocity_magnitude = allocate_2d_array(ny, nx);

    // Initialize obstacle mask
    for (int j = obstacle_y_start; j < obstacle_y_end; j++) {
        for (int i = obstacle_x_start; i < obstacle_x_end; i++) {
            h_mask[j][i] = 0;
        }
    }

    // Initialize flow: add inlet velocity from the left
    for (int j = 0; j < ny; j++) {
        h_u[j][0] = 1.0;
    }

    // Allocate device memory
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_u, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_v, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_p, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_div, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_u_next, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_v_next, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_u_center, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_v_center, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_velocity_magnitude, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_mask, ny*nx*s_i));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_converged, s_i));

    // Initialize device memory
    CUDA_CHECK_ERROR(hipMemset(d_u, 0, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_v, 0, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_p, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_div, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_u_next, 0, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_v_next, 0, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_u_center, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_v_center, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_velocity_magnitude, 0, ny*nx*s_d));

    // Copy data from host to device
    copy_host_to_device();

    // Define CUDA grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((nx + blockSize.x) / blockSize.x,
                  (ny + blockSize.y) / blockSize.y);

    // Open output files
    FILE *fp_u_center = fopen("u_center_data.bin", "wb");
    FILE *fp_v_center = fopen("v_center_data.bin", "wb");
    FILE *fp_magnitude = fopen("velocity_magnitude_data.bin", "wb");

    if (fp_u_center == NULL || fp_v_center == NULL || fp_magnitude == NULL) {
        perror("Error opening binary files");
        return 1;
    }

    int num_time_steps = (int)(total_time / dt);
    int output_interval = num_time_steps / max_number_of_frames;
    if (output_interval == 0) output_interval = 1;

    // Main time loop
    for (int t = 0; t < num_time_steps; t++) {
        clock_t step_start_time = clock();

        // Update velocities
        update_velocities_kernel<<<gridSize, blockSize>>>(
            d_u, d_v, d_p, d_u_next, d_v_next, nx, ny, dx, dy, dt, nu);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        // Copy updated velocities
        update_velocities_final_kernel<<<gridSize, blockSize>>>(
            d_u, d_v, d_u_next, d_v_next, nx, ny);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        // Pressure correction iterations
        for (int iter = 0; iter < max_iterations; iter++) {
            // Compute divergence
            compute_divergence_kernel<<<gridSize, blockSize>>>(d_u, d_v, d_div,
                                                               nx, ny, dx, dy);
            // CUDA_CHECK_ERROR(cudaGetLastError());
            CUDA_CHECK_ERROR(hipDeviceSynchronize());

            // Initialize convergence flag to 1
            int h_converged = 1;
            CUDA_CHECK_ERROR(hipMemcpy(d_converged, &h_converged, s_i,
                                        hipMemcpyHostToDevice));

            // Apply pressure correction
            pressure_correction_kernel<<<gridSize, blockSize>>>(
                d_u, d_v, d_p, d_div, d_mask, nx, ny, dx, dy, dt, beta,
                div_tolerance, d_converged);
            // CUDA_CHECK_ERROR(cudaGetLastError());
            CUDA_CHECK_ERROR(hipDeviceSynchronize());

            // Check if converged
            CUDA_CHECK_ERROR(hipMemcpy(&h_converged, d_converged, s_i,
                                        hipMemcpyDeviceToHost));

            if (h_converged) {
                break;
            }
        }

        // Apply boundary conditions
        apply_boundary_conditions_kernel<<<gridSize, blockSize>>>(
            d_u, d_v, d_mask, nx, ny);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        // Calculate center velocities and magnitude
        calculate_center_velocities_and_magnitude_kernel<<<gridSize,
                                                           blockSize>>>(
            d_u, d_v, d_u_center, d_v_center, d_velocity_magnitude, nx, ny);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        clock_t step_end_time = clock();
        total_compute_time_seconds +=
            (double)(step_end_time - step_start_time) / CLOCKS_PER_SEC;

        // Output results at specified intervals
        if (t % output_interval == 0) {
            // Copy results from device to host for output
            copy_device_to_host();

            // Write to binary files
            write_state_to_binary(fp_u_center, fp_v_center, fp_magnitude);
        }
    }

    // Copy final results from device to host
    copy_device_to_host();

    // Close output files
    fclose(fp_u_center);
    fclose(fp_v_center);
    fclose(fp_magnitude);

    // Calculate total elapsed time
    clock_t total_end_time = clock();
    double total_time_seconds =
        (double)(total_end_time - total_start_time) / CLOCKS_PER_SEC;

    // Write simulation metadata
    write_metadata_to_json("simulation_metadata.json",
                           total_compute_time_seconds, total_time_seconds);

    // Free device memory
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_p);
    hipFree(d_div);
    hipFree(d_u_next);
    hipFree(d_v_next);
    hipFree(d_u_center);
    hipFree(d_v_center);
    hipFree(d_velocity_magnitude);
    hipFree(d_mask);
    hipFree(d_converged);

    // Free host memory
    free_2d_array(h_u);
    free_2d_array(h_v);
    free_2d_array(h_p);
    free_2d_array(h_div);
    free_2d_int_array(h_mask);
    free_2d_array(h_u_center);
    free_2d_array(h_v_center);
    free_2d_array(h_velocity_magnitude);

    return 0;
}